#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>

#include "pa2.h"

__global__ void AATrans(mtxel *mtx, mtxel *dest, int dim, int blksize, int smsize)
{
	int t = (blockDim.x * blockIdx.x + threadIdx.x) * blksize;
	/* Calculate the column the thread is working in.
	 * We are only computing half the matrix,
	 * since the matrix is symmetric along the diagonal.
	 */
	int c = floor((1+2*dim-sqrtf(1+4*dim+4*dim*dim-8*t))/2);
	/* The row follows from the column */
	int r = t - c * dim + c * (c - 1) / 2 + c;
	
	DBGPRINT("Thread %d Initial Position: (%d, %d) with dim %d and blocksize %d\n", t, r, c, dim, blksize);
	/* Will be treated as mtxel rows[2 * blksize][dim] 
	 * The first blksize arrays are for the rows of the matrix at r
	 * The second blksize arrays are for the rows of the matrix at c
	 */
	extern __shared__ mtxel rowmem[];
	
	int currentcol = -1;;
	/* Compute A A^T */
	for(int i = 0; i < blksize; i++) {
		if(c >= 0 && c < dim &&
			 r >= 0 && r < dim) {
			dest[c * dim + r] = 0.0;
			for(int k = 0; k < dim; k++) {
				/* Move our current column into fast shared memory
				 * I assume the compiler is smart enough not to implement it in this fashion
				 */
				// if(c != currentcol)
				// 	rowmem[k] = mtx[c * dim + k];
				dest[c * dim + r] += mtx[r * dim + k] * mtx[c * dim + k];
			}
			DBGPRINT("t: %d, Pos: (%d, %d), value: %f\n", t, blksize, r, c, dest[c * dim + r]);
			dest[r * dim + c] = dest[c * dim + r];
			currentcol = c;
			r++;
			if(r >= dim) {
				c++;
				r = c;
			}
		}
	}
}

__global__ void AATransSmall(mtxel *mtx, mtxel *dest, int dim)
{
	/* Naive implementation. Rather slow, even with small matrices */
	int t = blockDim.x * blockIdx.x + threadIdx.x;
	int c = t / dim;
	int r = t % dim;
	if(c >= 0 && c < dim && r >= 0 && r < dim) {
		dest[c * dim + r] = 0.0;
		for(int k = 0; k < dim; k++)
			dest[c * dim + r] += mtx[r * dim + k] * mtx[c * dim + k];
	}
}

void computeCUDA(mtxel *hostmtx, mtxel *dest, int dim)
{
	if(dim == 1)
		return;
	mtxel *devmtx, *devdest;

	hipMalloc(&devmtx, sizeof(mtxel[dim * dim]));
	hipMalloc(&devdest, sizeof(mtxel[dim * dim]));
	if(!devmtx || !devdest)
		return;
	hipMemset(devdest, 0, sizeof(mtxel[dim * dim]));
	hipMemcpy(devmtx, hostmtx, sizeof(mtxel[dim * dim]), hipMemcpyHostToDevice);

	/* blksize is the number of rows and columns a thread works with */
	int blksize = 1;
	/* maxdim * (maxdim + 1) / 2 < 2^16, while anything greater is above 2^16
	 * This constraint exists because CUDA only supports up to 2^16 blocks
	 */
	const int maxthreads = 128;
	int threads = dim * (dim + 1) / 2;
	/* Now calculate the size of the blocks each thread works with,
	 * and add one extra thread, just in case
	 */
	while(threads > maxthreads) {
		blksize *= 2;
		threads /= 2;
	}
	threads++;

	/* The threads shared memory will consist of blksize rows
	 * So the total shared memory is dim * blksize
	 */
	AATrans <<< threads, 1, sizeof(mtxel[dim]) >>>
		(devmtx, devdest, dim, blksize, dim);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("CUDA Error %d: %s\n", err, hipGetErrorString(err));
	}

	hipMemcpy(dest, devdest, sizeof(mtxel[dim * dim]), hipMemcpyDeviceToHost);
	hipFree(devmtx);
	hipFree(devdest);
}

void checkCUBLAS(hipblasStatus_t err, char *event)
{
	switch(err) {
	case HIPBLAS_STATUS_SUCCESS:
		break;
	default:
		printf("Unknown error %d! %s\n", err, event);
	}
}

void computeCUBLAS(mtxel *mtx, mtxel *dest, int dim)
{
	hipblasStatus_t err;
	mtxel *devmtx1, *devdest;
	err = cublasAlloc(dim * dim, sizeof(mtxel), (void **)&devmtx1);
	checkCUBLAS(err, "Allocated dev matrix 1");
	err = cublasAlloc(dim * dim, sizeof(mtxel), (void **)&devdest);
	checkCUBLAS(err, "Allocated dev dest matrix");
	err = hipblasSetMatrix(dim, dim, sizeof(mtxel), (void *)mtx, dim, (void *)devmtx1, dim);
	checkCUBLAS(err, "Set dev matrix 1");

	hipblasDgemm('T', 'N', dim, dim, dim, 1.0,
		    devmtx1, dim, devmtx1, dim, 0.0, devdest, dim);

	err = cublasGetError();
	checkCUBLAS(err, "Multiplied matrix");
	err = hipblasGetMatrix(dim, dim, sizeof(mtxel), (void *)devdest, dim, dest, dim);
	checkCUBLAS(err, "Got matrix");
	cublasFree(devmtx1);
	cublasFree(devdest);
}

int initCUDA()
{
	/* Make certain we have a CUDA capable machine */
	int count = 0;
	hipGetDeviceCount(&count);
	if(count == 0) {
		return 1;
	}
	/* Find out some information about it.
	 * Require at least compute 2.0
	 */
	hipSetDevice(0);
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev, 0);
	if(dev.major < 2) {
		return 2;
	}
	/* Make a call to a CUDA function so initialization time
	 * isn't included in our computeCUDA time calculation
	 */
	void *mem = NULL;
	hipMalloc(&mem, 0);
	if(mem)
		hipFree(mem);

	/* Similarly for CUBLAS */
	cublasInit();
	return 0;
}

void shutdownCUDA()
{
	cublasShutdown();
}
