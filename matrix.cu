#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "pa2.h"

__global__ void AATrans(mtxel *mtx, mtxel *dest, int dim)
{
	int t = blockDim.x * blockIdx.x + threadIdx.x;
	/* Calculate the column the thread is working in.
	 * We are only computing half the matrix,
	 * since the matrix is symmetric along the diagonal.
	 */
	int c = floor((1+2*dim-sqrtf(1+4*dim+4*dim*dim-8*t))/2);
	/* The row follows from the column */
	int r = t - c * dim + c * (c - 1) / 2 + c;
	DBGPRINT("Dim: %d, Thread: %d, Row: %d, Column: %d\n", dim, t, r, c);
	if(c >= 0 && c < dim && r >= 0 && r < dim) {
		dest[c * dim + r] = 0.0;
		for(int k = 0; k < dim; k++)
			dest[c * dim + r] += mtx[r * dim + k] * mtx[c * dim + k];
		dest[r * dim + c] = dest[c * dim + r];
	}
}

void computeCUDA(mtxel *hostmtx, mtxel *dest, int dim)
{
	if(dim == 1)
		return;
	mtxel *devmtx, *devdest;
	hipMalloc(&devmtx, sizeof(mtxel[dim * dim]));
	hipMalloc(&devdest, sizeof(mtxel[dim * dim]));
	if(!devmtx || !devdest)
		return;
	hipMemcpy(devmtx, hostmtx, sizeof(mtxel[dim * dim]), hipMemcpyHostToDevice);
	hipMemset(devdest, 0.0, dim * dim);

	AATrans <<<dim * (dim + 1) / 2, 1>>> (devmtx, devdest, dim);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("CUDA Error %d: %s\n", err, hipGetErrorString(err));
	}

	hipMemcpy(dest, devdest, sizeof(mtxel[dim * dim]), hipMemcpyDeviceToHost);
	hipFree(devmtx);
	hipFree(devdest);
}

void checkCUBLAS(hipblasStatus_t err, char *event)
{
	switch(err) {
	case HIPBLAS_STATUS_SUCCESS:
		break;
	default:
		printf("Unknown error %d! %s\n", err, event);
	}
}

void computeCUBLAS(mtxel *mtx, mtxel *dest, int dim)
{
	hipblasStatus_t err;
	mtxel *devmtx1, *devdest;
	err = cublasAlloc(dim * dim, sizeof(mtxel), (void **)&devmtx1);
	checkCUBLAS(err, "Allocated dev matrix 1");
	err = cublasAlloc(dim * dim, sizeof(mtxel), (void **)&devdest);
	checkCUBLAS(err, "Allocated dev dest matrix");
	err = hipblasSetMatrix(dim, dim, sizeof(mtxel), (void *)mtx, dim, (void *)devmtx1, dim);
	checkCUBLAS(err, "Set dev matrix 1");

	hipblasSgemm('T', 'N', dim, dim, dim, 1.0,
		    devmtx1, dim, devmtx1, dim, 0.0, devdest, dim);

	err = cublasGetError();
	checkCUBLAS(err, "Multiplied matrix");
	err = hipblasGetMatrix(dim, dim, sizeof(mtxel), (void *)devdest, dim, dest, dim);
	checkCUBLAS(err, "Got matrix");
	cublasFree(devmtx1);
	cublasFree(devdest);
}

int initCUDA()
{
	/* Make certain we have a CUDA capable machine */
	int count = 0;
	hipGetDeviceCount(&count);
	if(count == 0) {
		return 1;
	}
	/* Find out some information about it.
	 * Require at least compute 2.0
	 */
	hipSetDevice(0);
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev, 0);
	if(dev.major < 2) {
		return 2;
	}
	cublasInit();
	return 0;
}

void shutdownCUDA()
{
	cublasShutdown();
	
}
